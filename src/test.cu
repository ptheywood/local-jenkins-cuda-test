#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"

#include "stdlib.h"
#include "stdio.h"
#include "time.h"
#include <typeinfo>
#include <vector>
#include <algorithm>

#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Lazy CUDA Error handling
static void HandleError(const char *file, int line, hipError_t status = hipGetLastError()) {
	if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess)
	{
		if (status == hipErrorUnknown)
		{
			printf("%s(%i) An Unknown CUDA Error Occurred :(\n", file, line);
			exit(1);
		}
		printf("%s(%i) CUDA Error Occurred;\n%s\n", file, line, hipGetErrorString(status));
		exit(1);
	}
}

#define CUDA_CALL( err ) (HandleError(__FILE__, __LINE__ , err))
#define CUDA_CHECK() (HandleError(__FILE__, __LINE__))


// Kernals

__global__ void setQuantities(
	unsigned int numInputs,
	unsigned int value,
	unsigned int * d_quantity
	){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	if (tid < numInputs){
		d_quantity[tid] = value;
	}

}


__global__ void atomicInc_kernel(
	unsigned int numIterations, 
	unsigned int numInputs, 
	float * d_probabilities, 
	unsigned int * d_quantity,
	unsigned int * d_count
){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid < numInputs){
		if(tid == 0){
			printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
		}
		for (int iteration = 0; iteration < numIterations; iteration++){
			// If a value is less than the probabiltiy, apply the min.

			unsigned int old = atomicInc(d_quantity + tid, MAX);

			// If old is MAX, could not increment.
			if(tid == 0){
				printf("tid %u: iter %d, old %u\n", tid, iteration, old );
			}
			if(old < MAX){
				d_count[tid]++;
			}
		}
	}
}

__global__ void atomicDec_kernel(
	unsigned int numIterations, 
	unsigned int numInputs, 
	float * d_probabilities, 
	unsigned int * d_quantity,
	unsigned int * d_count
){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid < numInputs){
		if(tid == 0){
			printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
		}		
		for (int iteration = 0; iteration < numIterations; iteration++){

			unsigned int old = atomicDec(d_quantity + tid, MAX);

			if(tid == 0){
				printf("tid %u: iter %d, old %u\n", tid, iteration, old );
			}

			// If old is not the maximum value, we have claimed a resource?
			if(old > 0){
				d_count[tid]++;
			}
		}
	}
}

__device__ unsigned int atomicIncCAS(unsigned int * address, unsigned int val){
	unsigned int old = *address;
	unsigned int assumed;
	do {
		assumed = old;
		old = atomicCAS(address, assumed, ((assumed >= val) ? 0 : (assumed+1)));
	} while (assumed != old);
	return old;
}
__device__ unsigned int atomicDecCAS(unsigned int * address, unsigned int val){
	unsigned int old = *address;
	unsigned int assumed;
	do {
		assumed = old;
		old = atomicCAS(address, assumed, (((assumed == 0) | (assumed > val)) ? val : (assumed-1)));
	} while (assumed != old);
	return old;
}

__device__ unsigned int atomicIncNoWrap(unsigned int * address, unsigned int val){
	unsigned int old = *address;
	unsigned int assumed;
	do {
		assumed = old;
		old = atomicCAS(address, assumed, ((assumed >= val) ? assumed : (assumed+1)));
	} while (assumed != old);
	return old;
}

__device__ unsigned int atomicDecNoWrap(unsigned int * address, unsigned int val){
	unsigned int old = *address;
	unsigned int assumed;
	do {
		assumed = old;
		old = atomicCAS(address, assumed, (((assumed == 0) | (assumed > val)) ? assumed : (assumed-1)));
	} while (assumed != old);
	return old;
}


__global__ void atomicIncNoWrap_kernel(
	unsigned int numIterations, 
	unsigned int numInputs, 
	float * d_probabilities, 
	unsigned int * d_quantity,
	unsigned int * d_count
){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid < numInputs){
		if(tid == 0){
			printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
		}
		for (int iteration = 0; iteration < numIterations; iteration++){
			// If a value is less than the probabiltiy, apply the min.

			unsigned int old = atomicIncNoWrap(d_quantity + tid, MAX);

			// If old is MAX, could not increment.
			if(tid == 0){
				printf("tid %u: iter %d, old %u\n", tid, iteration, old );
			}
			if(old < MAX){
				d_count[tid]++;
			}
		}
	}
}
__global__ void atomicDecNoWrap_kernel(
	unsigned int numIterations, 
	unsigned int numInputs, 
	float * d_probabilities, 
	unsigned int * d_quantity,
	unsigned int * d_count
){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid < numInputs){
		if(tid == 0){
			printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
		}		
		for (int iteration = 0; iteration < numIterations; iteration++){

			unsigned int old = atomicDecNoWrap(d_quantity + tid, MAX);

			if(tid == 0){
				printf("tid %u: iter %d, old %u\n", tid, iteration, old );
			}

			// If old is not the maximum value, we have claimed a resource?
			if(old > 0){
				d_count[tid]++;
			}
		}
	}
}



void generateInputData(unsigned int numInputs, unsigned long long int seed, float * d_data){
	hiprandGenerator_t rng = NULL;
	// Create RNG
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT); // @todo - hiprand error check
	// Seed the RNG
	hiprandSetPseudoRandomGeneratorSeed(rng, seed); // @todo - hiprand error check
	// Populate device array
	hiprandGenerateUniform(rng, d_data, numInputs); // @todo - hiprand error check
	// Cleanup rng
	hiprandDestroyGenerator(rng); // @todo - hiprand error check
}

void checkUsage(
	int argc,
	char *argv[],
	unsigned int *numRepeats,
	unsigned int *numIterations,
	unsigned int *numElements,
	unsigned long long int *seed,
	unsigned int *device
	){

		bool helpFlag = false;
		for(int i = 1; i < argc; i++){
			if(strcmp(argv[i], "-h") == 0){
				helpFlag = true;
			} else if(strcmp(argv[i], "--help") == 0){
				helpFlag = true;
			}
		}

		// If an incorrect number of arguments is specified, or -h is any arguement print usage.
		if (argc < MIN_ARGS || argc > MAX_ARGS || helpFlag ){
			const char *usage = "Usage: \n"
				"%s <num_iterations> <num_elements> <seed> <device>\n"
				"\n"
				"    <num_iterations> number of iterations to repeat (default %u)\n"
				"    <num_elements>   number of threads to launch (default %u)\n"
				"    <seed>           seed for RNG (default %llu)\n"
				"    <device>         CUDA Device index (default %d)\n"
				"\n";
			fprintf(stdout, usage, argv[ARG_EXECUTABLE], DEFAULT_NUM_ITERATIONS, DEFAULT_NUM_ELEMENTS, DEFAULT_SEED, DEFAULT_DEVICE);
			fflush(stdout);
			exit(EXIT_FAILURE);
		}

		// If there are more than 1 arg (the filename)5
		if(argc > MIN_ARGS){
			// Extract the number of repeats
			(*numRepeats) = (unsigned int) atoi(argv[ARG_REPEATS]);
			// Extract the number of iterations
			(*numIterations) = (unsigned int) atoi(argv[ARG_ITERATIONS]);
			// Extract the number of elements
			(*numElements) = (unsigned int) atoi(argv[ARG_ELEMENTS]);
			// Extract the seed
			(*seed) = strtoull(argv[ARG_SEED], nullptr, 0);
			if (argc >= ARG_DEVICE + 1){
				// Extract the device
				(*device) = (unsigned int)atoi(argv[ARG_DEVICE]);
			}

		}

		printf("repeats:    %u\n", (*numRepeats));
		printf("iterations: %u\n", (*numIterations));
		printf("threads:    %u\n", (*numElements));
		printf("seed:       %llu\n", (*seed));
		printf("device:     %u\n", (*device));

}

void initDevice(unsigned int device, int *major, int *minor){
	int deviceCount = 0;
	hipError_t status;
	// Get the number of cuda device.
	status = hipGetDeviceCount(&deviceCount);
	if (status != hipSuccess){
		fprintf(stderr, "Cuda Error getting device count.\n");
		fflush(stderr);
		exit(EXIT_FAILURE);
	}
	// If there are any devices
	if (deviceCount > 0){
		// Ensure the device count is not bad.
		if (device >= (unsigned int)deviceCount){
			device = DEFAULT_DEVICE;
			fprintf(stdout, "Warning: device %d is invalid, using device %d\n", device, DEFAULT_DEVICE);
			fflush(stdout);
		}
		// Set the device
		status = hipSetDevice(device);
		// If there were no errors, proceed.
		if (status == hipSuccess){
			// Get properties
			hipDeviceProp_t props;
			status = hipGetDeviceProperties(&props, device);
			// If we have properties, print the device.
			if (status == hipSuccess){
				fprintf(stdout, "Device: %s\n  pci %d bus %d\n  tcc %d\n  SM %d%d\n\n", props.name, props.pciDeviceID, props.pciBusID, props.tccDriver, props.major, props.minor);
				(*major) = props.major;
				(*minor) = props.minor;
			}
		}
		else {
			fprintf(stderr, "Error setting CUDA Device %d.\n", device);
			fflush(stderr);
			exit(EXIT_FAILURE);
		}
	}
	else {
		fprintf(stderr, "Error: No CUDA Device found.\n");
		fflush(stderr);
		exit(EXIT_FAILURE);
	}			
}

template <typename T, bool INC_NOT_DEC, bool NO_WRAP, bool verbose>
int test(
	unsigned int numRepeats, 
	unsigned int numIterations, 
	unsigned int numElements, 
	unsigned long long int seed, 
	float * d_probabilities,
	unsigned int * d_quantity, 
	unsigned int * d_count,
	unsigned int * h_quantity,
	unsigned int * h_count
	){
	unsigned int initialValue = 0;
	
	if (INC_NOT_DEC){
		if(NO_WRAP){
			fprintf(stdout, "atomicIncNoWrap \n");
		} else {
			fprintf(stdout, "atomicInc \n");
		}
		initialValue = 0;
	}
	else {
		if(NO_WRAP){
			fprintf(stdout, "atomicDecNoWrap \n");
		} else {
			fprintf(stdout, "atomicDec \n");
		}
		initialValue = MAX;
	}

	float milliTotal = 0.0f;
	int blockSize = 0;
	int minGridSize = 0;
	int gridSize = 0;
	for (unsigned int repeat = 0; repeat < numRepeats; repeat++){
		// Reset counts
		CUDA_CALL(hipMemset(d_count, 0, numElements * sizeof(unsigned int)));
		
		// REset quantities

		CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, setQuantities, 0, numElements));
		gridSize = (numElements + blockSize - 1) / blockSize;
		setQuantities << <gridSize, blockSize >> >(numElements, initialValue, d_quantity);
		CUDA_CHECK();
		

		// Create timing elements
		hipEvent_t start, stop;
		float milliseconds = 0;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// Get pointer to kernel
		void(*kernel)(unsigned int, unsigned int, float*, unsigned int *, unsigned int *);
		if (INC_NOT_DEC){
			if(NO_WRAP){
				kernel = atomicIncNoWrap_kernel;
			} else {
				kernel = atomicInc_kernel;
			}
		}
		else {
			if(NO_WRAP){
				kernel = atomicDecNoWrap_kernel;
			} else {
				kernel = atomicDec_kernel;
			}
		}

		// Compute launch args and launch kernel
		CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, numElements));
		gridSize = (numElements + blockSize - 1) / blockSize;

		// Execute the kernel
		CUDA_CALL(hipEventRecord(start));
		kernel << <gridSize, blockSize >> >(numIterations, numElements, d_probabilities, d_quantity, d_count);
		CUDA_CHECK();
		hipDeviceSynchronize();
		CUDA_CALL(hipEventRecord(stop));

		// Capture timing 
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);

		// Copy out results
		CUDA_CALL(hipMemcpy(h_count, d_count, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(h_quantity, d_quantity, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost));

		// Calculate some stats based on counts.
		// for(unsigned int i = 0; i < numElements; i++){
		for(unsigned int i = 0; i < 1; i++){
			fprintf(stdout, "%u: count %u, quantity %u\n", i, h_count[i], h_quantity[i]);
		}

		if(verbose){
			fprintf(stdout, "  > time %fms value ", milliseconds);
		}

		fflush(stdout);
		milliTotal += milliseconds;
	}

	float milliAverage = milliTotal / numRepeats;


	fprintf(stdout, "  Value: ");
	fprintf(stdout, "  Total  : %fms\n", milliTotal);
	fprintf(stdout, "  Average: %fms\n\n", milliAverage);
	fflush(stdout);


	// return milliTotal < 0.12f ? EXIT_SUCCESS : EXIT_FAILURE;
	return EXIT_SUCCESS;
}

int main(int argc, char *argv[])
{
	unsigned int numRepeats = DEFAULT_NUM_REPEATS;
	unsigned int numIterations = DEFAULT_NUM_ITERATIONS;
	unsigned int numElements = DEFAULT_NUM_ELEMENTS;
	unsigned long long int seed = DEFAULT_SEED;
	unsigned int device = DEFAULT_DEVICE;
	int major = 0;
	int minor = 0;

	checkUsage(argc, argv, &numRepeats, &numIterations, &numElements, &seed, &device);

	// Initialise the device
	initDevice(device, &major, &minor);
 
	// Alloc Rands.
	float *d_probabilities = NULL;
	CUDA_CALL(hipMalloc((void**)&d_probabilities, numElements * sizeof(float)));

	// Alloc quantity as unsigned int
	unsigned int *h_quantity = (unsigned int *)malloc(numElements * sizeof(unsigned int));
	unsigned int *d_quantity = NULL;
	CUDA_CALL(hipMalloc((void**)&d_quantity, numElements * sizeof(unsigned int)));
	
	unsigned int *h_count = (unsigned int *)malloc(numElements * sizeof(unsigned int));
	unsigned int *d_count = NULL;
	CUDA_CALL(hipMalloc((void**)&d_count, numElements * sizeof(unsigned int)));

	// Generate rands
	generateInputData(numElements, seed, d_probabilities);

	std::vector<int> testResults = std::vector<int>();

	// Test float atomicInc
	testResults.push_back(
		test<unsigned int, true, false, VERBOSE>(numRepeats, numIterations, numElements, seed, d_probabilities, d_quantity, d_count, h_quantity, h_count)
	);
	testResults.push_back(
		test<unsigned int, true, true, VERBOSE>(numRepeats, numIterations, numElements, seed, d_probabilities, d_quantity, d_count, h_quantity, h_count)
	);
	// Test float atomicDec
	testResults.push_back(
		test<unsigned int, false, false, VERBOSE>(numRepeats, numIterations, numElements, seed, d_probabilities, d_quantity, d_count, h_quantity, h_count)
	);
	testResults.push_back(
		test<unsigned int, false, true, VERBOSE>(numRepeats, numIterations, numElements, seed, d_probabilities, d_quantity, d_count, h_quantity, h_count)
	);


	size_t numPasses = std::count(testResults.begin(), testResults.end(), 0);
	size_t numTests = testResults.size();

	int retcode = numPasses == numTests ? EXIT_SUCCESS : EXIT_FAILURE;

	printf("testResults: %lu passes of %lu: returnCode %d\n", numPasses, numTests, retcode);



	// Free arrays.
	CUDA_CALL(hipFree(d_probabilities));
	CUDA_CALL(hipFree(d_quantity));
	CUDA_CALL(hipFree(d_count));
	free(h_count);
	free(h_quantity);


	// Reset the device.
	CUDA_CALL(hipDeviceReset());

    return retcode;
}
